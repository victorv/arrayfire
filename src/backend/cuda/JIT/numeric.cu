#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include "types.h"

template<typename T> __device__ T sign(T a) { return signbit(a); }

#define MATH_BASIC(fn, T)                       \
    __device__ T ___##fn(T a)                   \
    {                                           \
        return fn(a);                           \
    }                                           \


#define MATH_NOOP(fn, T)                        \
    __device__ T ___##fn(T a)                   \
    {                                           \
        return a;                               \
    }                                           \


#define MATH_CAST(fn, T, Tc)                    \
    __device__ T ___##fn(T a)                   \
    {                                           \
        return (T)fn((Tc)a);                    \
    }                                           \

MATH_BASIC(floor, float)
MATH_BASIC(floor, double)
MATH_NOOP(floor, int)
MATH_NOOP(floor, uint)
MATH_NOOP(floor, char)
MATH_NOOP(floor, uchar)
MATH_NOOP(floor, uintl)
MATH_NOOP(floor, intl)
MATH_NOOP(floor, ushort)
MATH_NOOP(floor, short)

MATH_BASIC(ceil, float)
MATH_BASIC(ceil, double)
MATH_NOOP(ceil, int)
MATH_NOOP(ceil, uint)
MATH_NOOP(ceil, char)
MATH_NOOP(ceil, uchar)
MATH_NOOP(ceil, uintl)
MATH_NOOP(ceil, intl)
MATH_NOOP(ceil, ushort)
MATH_NOOP(ceil, short)

MATH_BASIC(round, float)
MATH_BASIC(round, double)
MATH_NOOP(round, int)
MATH_NOOP(round, uint)
MATH_NOOP(round, char)
MATH_NOOP(round, uchar)
MATH_NOOP(round, uintl)
MATH_NOOP(round, intl)
MATH_NOOP(round, ushort)
MATH_NOOP(round, short)

MATH_BASIC(trunc, float)
MATH_BASIC(trunc, double)
MATH_NOOP(trunc, int)
MATH_NOOP(trunc, uint)
MATH_NOOP(trunc, char)
MATH_NOOP(trunc, uchar)
MATH_NOOP(trunc, uintl)
MATH_NOOP(trunc, intl)
MATH_NOOP(trunc, ushort)
MATH_NOOP(trunc, short)

MATH_BASIC(sign, float)
MATH_BASIC(sign, double)
MATH_NOOP(sign, int)
MATH_NOOP(sign, uint)
MATH_NOOP(sign, char)
MATH_NOOP(sign, uchar)
MATH_NOOP(sign, uintl)
MATH_NOOP(sign, intl)
MATH_NOOP(sign, ushort)
MATH_NOOP(sign, short)

MATH_BASIC(abs, float)
MATH_BASIC(abs, double)
MATH_BASIC(abs, int)
MATH_CAST(abs, char, int)
MATH_NOOP(abs, uint)
MATH_NOOP(abs, uchar)
MATH_NOOP(abs, uintl)
MATH_NOOP(abs, intl)
MATH_NOOP(abs, ushort)
MATH_NOOP(abs, short)

MATH_BASIC(tgamma, float)
MATH_BASIC(tgamma, double)
MATH_CAST(tgamma, int   , float)
MATH_CAST(tgamma, uint  , float)
MATH_CAST(tgamma, char  , float)
MATH_CAST(tgamma, uchar , float)
MATH_CAST(tgamma, uintl , float)
MATH_CAST(tgamma, intl  , float)
MATH_CAST(tgamma, ushort, float)
MATH_CAST(tgamma, short , float)

MATH_BASIC(lgamma, float)
MATH_BASIC(lgamma, double)
MATH_CAST(lgamma, int   , float)
MATH_CAST(lgamma, uint  , float)
MATH_CAST(lgamma, char  , float)
MATH_CAST(lgamma, uchar , float)
MATH_CAST(lgamma, uintl , float)
MATH_CAST(lgamma, intl  , float)
MATH_CAST(lgamma, ushort, float)
MATH_CAST(lgamma, short , float)

MATH_NOOP(noop, float)
MATH_NOOP(noop, double)
MATH_NOOP(noop, cfloat)
MATH_NOOP(noop, cdouble)
MATH_NOOP(noop, int)
MATH_NOOP(noop, uint)
MATH_NOOP(noop, char)
MATH_NOOP(noop, uchar)
MATH_NOOP(noop, uintl)
MATH_NOOP(noop, intl)
MATH_NOOP(noop, ushort)
MATH_NOOP(noop, short)

__device__ float ___abs(cfloat a) { return hipCabsf(a); }
__device__ double ___abs(cdouble a) { return hipCabs(a); }

template<typename T> __device__ T rem(T a, T b) { return a % b; }
__device__ float rem(float a, float b) { return remainderf(a, b); }
__device__ double rem(double a, double b) { return remainder(a, b); }

template<typename T> __device__ T mod(T a, T b) { return a % b; }
__device__ float mod(float a, float b) { return fmodf(a, b); }
__device__ double mod(double a, double b) { return fmod(a, b); }

#define MATH2_BASIC(fn, T)                      \
    __device__ T ___##fn(T a, T b)              \
    {                                           \
        return fn(a, b);                        \
    }                                           \

#define MATH2(fn)                                   \
    MATH2_BASIC(fn, float)                          \
    MATH2_BASIC(fn, int)                            \
    MATH2_BASIC(fn, uint)                           \
    MATH2_BASIC(fn, intl)                           \
    MATH2_BASIC(fn, uintl)                          \
    MATH2_BASIC(fn, char)                           \
    MATH2_BASIC(fn, uchar)                          \
    MATH2_BASIC(fn, short)                          \
    MATH2_BASIC(fn, ushort)                         \
    __device__ double ___##fn(double a, double b)   \
    {                                               \
        return fn(a, b);                            \
    }                                               \

MATH2(min)
MATH2(max)
MATH2(mod)
MATH2(rem)

__device__ float ___hypot(float a, float b)
{
    return hypot(a, b);
}

__device__ double ___hypot(double a, double b)
{
    return hypot(a, b);
}

#define COMPARE_CPLX(fn, op, T)                 \
    __device__ T ___##fn(T a, T b)              \
    {                                           \
        return cabs2(a) op cabs2(b) ? a : b;    \
    }                                           \

COMPARE_CPLX(min, <, cfloat)
COMPARE_CPLX(min, <, cdouble)
COMPARE_CPLX(max, >, cfloat)
COMPARE_CPLX(max, >, cdouble)
