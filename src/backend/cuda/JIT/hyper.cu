#include "hip/hip_runtime.h"
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include "types.h"

#define MATH_BASIC(fn, T)                       \
    __device__ T ___##fn(T a)                   \
    {                                           \
        return fn##f((float)a);                 \
    }                                           \


#define MATH(fn)                                \
    MATH_BASIC(fn, float)                       \
    MATH_BASIC(fn, int)                         \
    MATH_BASIC(fn, uint)                        \
    MATH_BASIC(fn, char)                        \
    MATH_BASIC(fn, uchar)                       \
    MATH_BASIC(fn, uintl)                       \
    MATH_BASIC(fn, intl)                        \
    MATH_BASIC(fn, ushort)                      \
    MATH_BASIC(fn, short)                       \
    __device__ double ___##fn(double a)         \
    {                                           \
        return fn(a);                           \
    }                                           \


MATH(sinh)
MATH(cosh)
MATH(tanh)

MATH(asinh)
MATH(acosh)
MATH(atanh)
