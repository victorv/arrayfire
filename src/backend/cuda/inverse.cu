/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#include <inverse.hpp>
#include <err_common.hpp>

#if defined(WITH_CUDA_LINEAR_ALGEBRA)

#include <solve.hpp>
#include <identity.hpp>

namespace cuda
{

template<typename T>
Array<T> inverse(const Array<T> &in)
{
    Array<T> I = identity<T>(in.dims());
    return solve<T>(in, I);
}

#define INSTANTIATE(T)                                                                   \
    template Array<T> inverse<T> (const Array<T> &in);

INSTANTIATE(float)
INSTANTIATE(cfloat)
INSTANTIATE(double)
INSTANTIATE(cdouble)

}

#elif defined(WITH_CPU_LINEAR_ALGEBRA)
#include <cpu_lapack/cpu_inverse.hpp>

namespace cuda
{

template<typename T>
Array<T> inverse(const Array<T> &in)
{
    return cpu::inverse(in);
}

#define INSTANTIATE(T)                                                                   \
    template Array<T> inverse<T> (const Array<T> &in);

INSTANTIATE(float)
INSTANTIATE(cfloat)
INSTANTIATE(double)
INSTANTIATE(cdouble)

}

#else
namespace cuda
{

template<typename T>
Array<T> inverse(const Array<T> &in)
{
    AF_ERROR("CUDA cusolver not available. Linear Algebra is disabled",
              AF_ERR_NOT_CONFIGURED);
}

#define INSTANTIATE(T)                                                                   \
    template Array<T> inverse<T> (const Array<T> &in);

INSTANTIATE(float)
INSTANTIATE(cfloat)
INSTANTIATE(double)
INSTANTIATE(cdouble)

}

#endif
